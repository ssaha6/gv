
#include <hip/hip_runtime.h>
//pass
//--gridDim=[32768,1,1]    --blockDim=[512,1,1]

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=0; i<num_iterations; i++)
    {
        g_data[idx] += *factor;    // non-coalesced on purpose, to burn time
    }
}
